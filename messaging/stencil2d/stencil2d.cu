#include "hip/hip_runtime.h"
#include "hapi.h"
#include "stencil2d.h"

#define TILE_SIZE 16

__global__ void initKernel(double* temperature, double val, int block_x,
    int block_y, int thread_size) {
  int i_start = (blockDim.x * blockIdx.x + threadIdx.x) * thread_size + 1;
  int i_finish =
      (blockDim.x * blockIdx.x + threadIdx.x) * thread_size + thread_size;
  int j_start = (blockDim.y * blockIdx.y + threadIdx.y) * thread_size + 1;
  int j_finish =
      (blockDim.y * blockIdx.y + threadIdx.y) * thread_size + thread_size;

  for (int i = i_start; i <= i_finish; i++) {
    for (int j = j_start; j <= j_finish; j++) {
      if (i <= block_x && j <= block_y) {
        temperature[(block_x + 2) * j + i] = val;
      }
    }
  }
}

__global__ void packingKernel(double* temperature, double* west_ghost,
    double* east_ghost, double* north_ghost, double* south_ghost, int block_x,
    int block_y) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < block_y) {
    if (west_ghost) {
      west_ghost[i] = temperature[(block_x + 2) * (1 + i) + 1];
    }
  }
  else if (i < 2 * block_y) {
    if (east_ghost) {
      i -= block_y;
      east_ghost[i] = temperature[(block_x + 2) * (1 + i) + block_x];
    }
  }
  else if (i < 2 * block_y + block_x) {
    if (north_ghost) {
      i -= 2 * block_y;
      north_ghost[i] = temperature[(block_x + 2) + (1 + i)];
    }
  }
  else if (i < 2 * block_y + 2 * block_x) {
    if (south_ghost) {
      i -= (2 * block_y + block_x);
      south_ghost[i] = temperature[(block_x + 2) * block_y + (1 + i)];
    }
  }
}

__global__ void unpackingKernel(double* temperature, double* ghost, int width,
    int dir, int block_x, int block_y) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < width) {
    if (dir == WEST) {
      temperature[(block_x + 2) * (1 + i)] = ghost[i];
    }
    else if (dir == EAST) {
      temperature[(block_x + 2) * (1 + i) + (block_x + 1)] = ghost[i];
    }
    else if (dir == NORTH) {
      temperature[1 + i] = ghost[i];
    }
    else if (dir == SOUTH) {
      temperature[(block_x + 2) * (block_y + 1) + (1 + i)] = ghost[i];
    }
  }
}

__global__ void boundaryKernel(double* temperature, bool west_bound, bool east_bound,
    bool north_bound, bool south_bound, int block_x, int block_y) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < block_y) {
    if (west_bound) {
      temperature[(block_x + 2) * (1 + i)] = 1.0;
    }
  }
  else if (i < 2 * block_y) {
    if (east_bound) {
      i -= block_y;
      temperature[(block_x + 2) * (1 + i) + (block_x + 1)] = 1.0;
    }
  }
  else if (i < 2 * block_y + block_x) {
    if (north_bound) {
      i -= 2 * block_y;
      temperature[1 + i] = 1.0;
    }
  }
  else if (i < 2 * block_y + 2 * block_x) {
    if (south_bound) {
      i -= (2 * block_y + block_x);
      temperature[(block_x + 2) * (block_y + 1) + (1 + i)] = 1.0;
    }
  }
}

__global__ void stencilKernel(double* temperature, double* new_temperature,
    int block_x, int block_y, int thread_size) {
  int i_start = (blockDim.x * blockIdx.x + threadIdx.x) * thread_size + 1;
  int i_finish =
      (blockDim.x * blockIdx.x + threadIdx.x) * thread_size + thread_size;
  int j_start = (blockDim.y * blockIdx.y + threadIdx.y) * thread_size + 1;
  int j_finish =
      (blockDim.y * blockIdx.y + threadIdx.y) * thread_size + thread_size;

  for (int i = i_start; i <= i_finish; i++) {
    for (int j = j_start; j <= j_finish; j++) {
      if (i <= block_x && j <= block_y) {
        new_temperature[j * (block_x + 2) + i] =
            (temperature[j * (block_x + 2) + (i - 1)] +
             temperature[j * (block_x + 2) + (i + 1)] +
             temperature[(j - 1) * (block_x + 2) + i] +
             temperature[(j + 1) * (block_x + 2) + i] +
             temperature[j * (block_x + 2) + i]) *
            DIVIDEBY5;
      }
    }
  }
}

void invokeInitKernel(double* temperature, double val, int block_x, int block_y,
    int thread_size, hipStream_t stream) {
  dim3 block_dim(TILE_SIZE, TILE_SIZE);
  dim3 grid_dim(
      (block_x + (block_dim.x * thread_size - 1)) / (block_dim.x * thread_size),
      (block_y + (block_dim.y * thread_size - 1)) / (block_dim.y * thread_size));

  initKernel<<<grid_dim, block_dim, 0, stream>>>(temperature, val, block_x,
      block_y, thread_size);

  hapiCheck(hipPeekAtLastError());
}

void invokePackingKernel(double* temperature, double* west_ghost, double* east_ghost,
    double* north_ghost, double* south_ghost, int block_x, int block_y,
    hipStream_t stream) {
  dim3 block_dim(TILE_SIZE * TILE_SIZE);
  dim3 grid_dim((2 * block_x + 2 * block_y + block_dim.x - 1) / block_dim.x);

  packingKernel<<<grid_dim, block_dim, 0, stream>>>(temperature, west_ghost,
      east_ghost, north_ghost, south_ghost, block_x, block_y);

  hapiCheck(hipPeekAtLastError());
}

void invokeUnpackingKernel(double* temperature, double* ghost, int width,
    int dir, int block_x, int block_y, hipStream_t stream) {
  dim3 block_dim(TILE_SIZE);
  dim3 grid_dim((width + block_dim.x - 1) / block_dim.x);

  unpackingKernel<<<grid_dim, block_dim, 0, stream>>>(temperature, ghost, width,
      dir, block_x, block_y);

  hapiCheck(hipPeekAtLastError());
}

void invokeBoundaryKernel(double* temperature, bool west_bound, bool east_bound,
    bool north_bound, bool south_bound, int block_x, int block_y,
    hipStream_t stream) {
  dim3 block_dim(TILE_SIZE * TILE_SIZE);
  dim3 grid_dim((2 * block_x + 2 * block_y + block_dim.x - 1) / block_dim.x);

  boundaryKernel<<<grid_dim, block_dim, 0, stream>>>(temperature, west_bound,
      east_bound, north_bound, south_bound, block_x, block_y);

  hapiCheck(hipPeekAtLastError());
}

void invokeStencilKernel(double* d_temperature, double* d_new_temperature,
    int block_x, int block_y, int thread_size, hipStream_t stream) {
  dim3 block_dim(TILE_SIZE, TILE_SIZE);
  dim3 grid_dim(
      (block_x + (block_dim.x * thread_size - 1)) / (block_dim.x * thread_size),
      (block_y + (block_dim.y * thread_size - 1)) / (block_dim.y * thread_size));

  stencilKernel<<<grid_dim, block_dim, 0, stream>>>(d_temperature,
      d_new_temperature, block_x, block_y, thread_size);

  hapiCheck(hipPeekAtLastError());
}
